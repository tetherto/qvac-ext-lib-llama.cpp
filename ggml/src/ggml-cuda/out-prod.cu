#include "out-prod.cuh"
#include "convert.cuh"

#include <cstdint>

void ggml_cuda_out_prod(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_TENSOR_BINARY_OP_LOCALS

    const bool src0_is_quantized = (src0->type != GGML_TYPE_F32 && src0->type != GGML_TYPE_F16);
    const bool src1_is_quantized = (src1->type != GGML_TYPE_F32 && src1->type != GGML_TYPE_F16);

    // if (src0_is_quantized || src1_is_quantized) {
    //     printf("DEBUG: OUT_PROD with quantized tensors - src0_quantized=%d, src1_quantized=%d\n", 
    //            src0_is_quantized, src1_is_quantized);
    //     fflush(stdout);
    // }

    // GGML_ASSERT(src0->type == GGML_TYPE_F32);
    // GGML_ASSERT(src1->type == GGML_TYPE_F32);

    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    // temp buffers
    float * src0_f32 = nullptr;
    float * src1_f32 = nullptr;
    bool allocated_src0 = false;
    bool allocated_src1 = false;
    hipStream_t   stream = ctx.stream();

    if (src0_is_quantized) {
        const size_t src0_size = ggml_nelements(src0) * sizeof(float);
        CUDA_CHECK(hipMallocAsync(&src0_f32, src0_size, stream));
        allocated_src0 = true;

        // Dequantize
        auto dequantize_fn = ggml_get_to_fp32_cuda(src0->type);
        if (dequantize_fn) {
            dequantize_fn(src0->data, src0_f32, ggml_nelements(src0), stream);
        } else {
            CUDA_CHECK(hipFreeAsync(src0_f32, stream));
            GGML_ABORT("Unsupported quant type for src0");
        }
    } else {
        src0_f32 = (float *) src0->data;
    } 

    if (src1_is_quantized) {
        const size_t src1_size = ggml_nelements(src1) * sizeof(float);
        CUDA_CHECK(hipMallocAsync(&src1_f32, src1_size, stream));
        allocated_src1 = true;

        auto dequantize_fn = ggml_get_to_fp32_cuda(src1->type);
        if (dequantize_fn) {
            dequantize_fn(src1->data, src1_f32, ggml_nelements(src0), stream);
        } else {
            CUDA_CHECK(hipFreeAsync(src1_f32, stream));
            GGML_ABORT("Unsupported quant type for src1");
        }
    } else {
        src1_f32 = (float *) src1->data;
    } 
    

    GGML_ASSERT(ne01 == ne11);
    GGML_ASSERT(ne0 == ne00);
    GGML_ASSERT(ne1 == ne10);

    GGML_ASSERT(ne2 % src0->ne[2] == 0);
    GGML_ASSERT(ne3 % src0->ne[3] == 0);

    GGML_ASSERT(ne2 == src1->ne[2]);
    GGML_ASSERT(ne3 == src1->ne[3]);

    // const float * src0_d = (const float *) src0->data;
    // const float * src1_d = (const float *) src1->data;

    // Use dequantized data
    const float * src0_d = src0_f32;
    const float * src1_d = src1_f32;
    float       *  dst_d = (float       *)  dst->data;

    hipblasHandle_t handle = ctx.cublas_handle();

    const float alpha = 1.0f;
    const float beta = 0.0f;

    CUBLAS_CHECK(hipblasSetStream(handle, stream));

    // const int64_t lda = nb01 / sizeof(float);
    const int64_t lda = allocated_src0 ? ne00 : (nb01 / sizeof(float));
    const int64_t ldc = nb1  / sizeof(float);

    const bool src1_T = ggml_is_transposed(src1);
    const hipblasOperation_t src1_cublas_op =  src1_T ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    // const int64_t           ldb            = (src1_T ?        nb10 :        nb11) /  sizeof(float);
    const int64_t           ldb            = allocated_src1 ? 
                                             (src1_T ? ne10 : ne11) :
                                             ((src1_T ?        nb10 :        nb11) /  sizeof(float));
                                
    // GGML_ASSERT(                             (src1_T ?        nb11 :        nb10) == sizeof(float));
    // Only assert for non dequantized src1
    if (!allocated_src1) {
        GGML_ASSERT((src1_T ? nb11 : nb10) == sizeof(float));
    }

    // data strides in dimensions 2/3
    // const size_t s02 = nb02 / sizeof(float);
    // const size_t s03 = nb03 / sizeof(float);
    // const size_t s12 = nb12 / sizeof(float);
    // const size_t s13 = nb13 / sizeof(float);
    const size_t s02 = allocated_src0 ? (ne00 * ne01) : nb02 / sizeof(float);
    const size_t s03 = allocated_src0 ? (ne00 * ne01 * ne02): nb03 / sizeof(float);
    const size_t s12 = allocated_src1 ? (ne10 * ne11) :  nb12 / sizeof(float);
    const size_t s13 = allocated_src1 ? (ne10 * ne11 * ne12) : nb13 / sizeof(float);
    const size_t s2  = nb2  / sizeof(float);
    const size_t s3  = nb3  / sizeof(float);

    // dps == dst per src0, used for group query attention
    const int64_t dps2 = ne2 / ne02;
    const int64_t dps3 = ne3 / ne03;

    // TODO batched matrix multiplication
    for (int64_t i3 = 0; i3 < ne3; ++i3) {
        for (int64_t i2 = 0; i2 < ne2; ++i2) {
            CUBLAS_CHECK(
                hipblasSgemm(handle, HIPBLAS_OP_N, src1_cublas_op,
                        ne0, ne1, ne01,
                        &alpha, src0_d + (i3/dps3)*s03 + (i2/dps2)*s02, lda,
                                src1_d +  i3      *s13 +  i2      *s12, ldb,
                        &beta,  dst_d  +  i3      *s3  +  i2      *s2,  ldc));
        }
    }

    if (allocated_src0) {
        CUDA_CHECK(hipFreeAsync(src0_f32, stream));
        // printf("DEBUG: Freed dequantized src0 buffer\n");
    }
    if (allocated_src1) {
        CUDA_CHECK(hipFreeAsync(src1_f32, stream));
        // // printf("DEBUG: Freed dequantized src1 buffer\n");
    }
    
    // printf("DEBUG: CUDA OUT_PROD completed successfully\n");
    fflush(stdout);
}
